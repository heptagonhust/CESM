
#include <hip/hip_runtime.h>
#include <stdio.h>

int my_task;
int cuda_initialized = 0;

//Fortran entry for initializing CUDA
extern "C"
void cuda_init_(int *pmy_task) {
  if (cuda_initialized == 0) {
    cuda_initialized = 1;
    my_task = *pmy_task;
    int deviceCount = 0;

    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) fprintf(stderr, "Error in cuda initialization: %s\n", hipGetErrorString( err ));

    if (deviceCount < 1) {
      fprintf(stderr,"Error: less than 1 cuda capable device detected proc=%d\n", my_task);
    }

    int dev = my_task % deviceCount;
    //fprintf(stdout,"Process %d: using CUDA device %d\n",my_task,dev);

    hipSetDeviceFlags(hipDeviceMapHost);
    hipSetDevice(dev);

    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSynchronize();
  }
}

extern "C"
//Fortran entry for allocating pinned memory
void cudamallochost_(void **hostptr, int *p_size) {
  if (!cuda_initialized) {
    printf("Error: cudamallochost called before cuda_init\n");
  }
 
  hipError_t err;

  err = hipHostAlloc((void **)hostptr, (*p_size)*sizeof(double), hipHostMallocMapped);
  if (err != hipSuccess) fprintf(stderr, "Error in cudaHostAlloc: %s\n", hipGetErrorString( err ));
}