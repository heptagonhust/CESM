//注意r8是double精度 我的数学公式转换没错吧？ 帮我检查一下？
//所有的.cu都要用*_cuda.cu 结尾，因为我的编译脚本就是这么判断的
//0.00000013816，和 8.1328e-3_r8 小数点直接这样转换，应该没问题吧，，

#include <hip/hip_runtime.h>
#include <math.h>   
extern "C" 

//因为用了interface，注明了是c ，所以这里不用在函数名后面加上 '_'
void svp_water_cuda(double* ptboil,double* pt,double* es){
   //Y00: use cuda to do this  
   double tboil=*ptboil;
   double t=*pt;
     double tmp=(-7.90298*(tboil/t-1.0)+ \
       5.02808*log10(tboil/t)- \
       0.00000013816*(exp10(11.344*(1.0-t/tboil))-1.0)+ \
       0.0081328*(exp10(-3.49149*(tboil/t-1.0))-1.0)+ \
       log10(1013.246));
  *es= exp10(tmp)*100.0;

}