
#include <hip/hip_runtime.h>
__global__
void cudaCal(double* pk,double pe,double akap){
   *pk=pow(pe,akap);
}
//akap 每轮计算一直是一个固定的，考虑stream一下？

extern "C" 
void calpk_(double* pk,double* pe,double* akap){
   cudaCal<<<1,1>>>(pk,*pe,*akap);
   hipDeviceSynchronize();
}

extern "C" //fortran直接调用的函数名一定要小写
void calpkcuda_(double* pk,double* pe,double* akap,int* km, int* i1, int* i2,int* jfirst,int* jp,double* ptop){

}
