
#include <hip/hip_runtime.h>
//直接把那个循环里的openmp移除？ 这样方便cuda计算？？

__global__
void cudaCal(double* pk,double pe,double akap){
   *pk=pow(pe,akap);
}
//akap 每轮计算一直是一个固定的，考虑stream一下？

extern "C" //方便gcc/g++ fortran链接
void calpk_(double* pk,double* pe,double* akap){
   cudaCal<<<1,1>>>(pk,*pe,*akap);
   hipDeviceSynchronize();
}

//必须小写！！！
void calpkCuda_(double*pk,double*pe,double*akap,int* km, int* i1, int* i2,int* j){
   for(int z=*i1;z < *i2;z++){
      for(int x=0;x<*km+1;x++){
         for(int y=0;y<*j;y++){//TODO 这个循环范围还有问题，然后fortran和c的矩阵还不一样, 我还是一个个做吧，，，
         cudaCal<<<1,1>>>(pk,*pe,*akap);
         }
      }
   }
}