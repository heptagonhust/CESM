#include "hip/hip_runtime.h"
//只是放在这里，还没有研究fft的输入输出
//如果需要单独把 init 函数和 destroy 函数抽出来，我再到调用的地方加
//https://github.com/qzwlecr/CESM/issues/12
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <fstream>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>
#include <string>
#include <unordered_map>

#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
using namespace std::chrono;

#define LOG(arg) printf("%s=%d, ", #arg, (arg))

using real_t = hipfftDoubleReal;
using complex_t = hipfftDoubleComplex;
extern "C" void cuda_fft991_batch_host_(       //
    int* batch_size_, int* batch_distance_,    //
    double* a_,                                // inout, elements[lot][N+2]
    int* inc_,                                 // data memory addr increment of elements
    int* jump_,                                // data memory addr increment of vector

    int* n_,       // count of elements in a vector
    int* lot_,     // count of vectors
    int* ISIGN_    // -1 => time2freq, +1 => freq2time
) {
    // to be discard
    auto batch_size = *batch_size_;
    auto batch_distance = *batch_distance_;
    double* dev_a;
    int data_byte = batch_distance * batch_size * sizeof(double);
    hipMalloc(&dev_a, data_byte);
    hipMemcpy(dev_a, a_, data_byte, hipMemcpyHostToDevice);
    if(batch_distance == *jump_ * *lot_) {
        int fake_lot = *lot_ * batch_size;
        cuda_fft991_(dev_a, inc_, jump_, n_, &fake_lot, ISIGN_);
    } else {
        assert(false);
    }
    hipMemcpy(a_, dev_a, data_byte, hipMemcpyDeviceToHost);
    hipFree(dev_a);
}

extern "C" void cuda_fft991_(    //
    double* a_,                  // inout, elements[lot][N+2]
    int* inc_,                   // data memory addr increment of elements
    int* jump_,                  // data memory addr increment of vector
    int* n_,                     // count of elements in a vector
    int* lot_,                   // count of vectors
    int* ISIGN_                  // -1 => time2freq, +1 => freq2time

) {
    thread_local std::unordered_map<int, std::pair<hipfftHandle, hipfftHandle> > record;
    int batch_count = *lot_;
    assert(*jump_ == 146);
    int total_count = 146 * batch_count;
    if(record.count(batch_count) == 0) {
        auto& fwd_plan = record[batch_count].first;
        auto& bck_plan = record[batch_count].second;
        int n = *n_;
        int stride = *inc_;
        int real_dist = *jump_;
        int complex_dist = real_dist / 2;
        int ranks[] = {n};
        int new_count = batch_count * real_dist;
        // assert(!total_count || new_count == total_count);
        total_count = new_count;
        printf("\n<<<<");
        LOG(n);
        LOG(stride);
        LOG(real_dist);
        LOG(batch_count);
        printf("\n");
        hipfftPlanMany(&fwd_plan, 1, ranks, nullptr, stride, real_dist, nullptr, stride,
                      complex_dist, HIPFFT_D2Z, batch_count);
        hipfftPlanMany(&bck_plan, 1, ranks, nullptr, stride, complex_dist, nullptr, stride,
                      real_dist, HIPFFT_Z2D, batch_count);
    }

    auto& fwd_plan = record[batch_count].first;
    auto& bck_plan = record[batch_count].second;
    if(*ISIGN_ == -1) {
        // fwd_plan
        auto status = hipfftExecD2Z(fwd_plan, (real_t*)a_, (complex_t*)a_);
        // std::cout << "total_count=" << total_count << std::endl;
        // auto fn =;
        thrust::transform(thrust::system::cuda::par, a_, a_ + total_count, a_,
                          [=] __device__(double x) { return x / 144; });
        // printf("{executed=%d}", status);
    } else {
        hipfftExecZ2D(bck_plan, (complex_t*)a_, (real_t*)a_);
    }
}

extern "C" void needle_(                      //
    double* a_,                               // inout, elements[lot][N+2]
    int* batch_size_, int* batch_distance_    // distance
) {
    static int counter = 0;
    printf("[needle at %p, size=%d, dist=%d]", a_, *batch_size_, *batch_distance_);
    ++counter;
    auto current_time = system_clock::now();
    auto path = "/home/mike/tmp/";
    auto filename = path + std::to_string(system_clock::to_time_t(current_time)) + "-" +
                    std::to_string(counter) + ".dat";
    int data_byte = *batch_distance_ * *batch_size_ * sizeof(double);
    std::ofstream fout(filename, std::ios::binary);
    fout.write((char*)a_, data_byte);
}

template <typename T, bool is_managed = false>
T* cuda_alloc(int size) {
    int bytes = sizeof(T) * size : T * tmp;
    if(is_managed) {
        hipMallocManaged(&tmp, bytes);
    } else {
        hipMalloc(&tmp, bytes);
    }
    return tmp;
}

// blocks: y_dim
// threads: x_dim
// s_ should be constant array

// need adjustment
constexpr int MAX_S_SIZE = 128 + 16;
struct PftRecord {
    // here is the wtf
    int s_size;
    int x_dim;
    double s_rev[MAX_S_SIZE];
    int fft_count;
    int encode_ids[MAX_S_SIZE];
    int decode_ids[MAX_S_SIZE];
    int fwd_plan, bck_plan;
    double* dev_damp;                // of fft_count * (x_dim + 2), keep it in memory
    hipfftDoubleReal* dev_origin;     // of fft_count * (x_dim), keep it in memory
    hipfftDoubleComplex* dev_freq;    // of fft_count * (x_dim + 2), keep it in memory
    double* dev_inout;               // of s_size * x_dim
};

static PftRecord pft_records[4] = {};
static __constant__ PftRecord dev_pft_records[4];

extern "C"    //
    void
    cuda_pft_cf_record_(int* plan_id_, double* s_, int* s_beg_, int* s_end_,
                        double* damp_, int* im_, int* fft_flt_) {
    // initalize this region for further call
    int plan_id = *plan_id_;
    assert(plan_id >= 0 && plan_id < 4);
    int s_size = *s_end_ - *s_beg_ + 1;
    assert(s_size < MAX_S_SIZE);
    int x_dim = *im_;
    auto dev_record_ptr = &dev_pft_records[plan_id];
    auto& record = pft_records[plan_id];
    auto* encode_ids = record.encode_ids;
    auto* decode_ids = record.decode_ids;

    int fft_count = 0;

    bool force_fft = (bool)*fft_flt_;
    for(int i = 0; i < s_size; ++i) {
        auto coef = s_[i];
        if(coef <= 1.01) {
            // skip
            encode_ids[i] = -2;
        } else if(!force_fft && coef <= 4.0) {
            // shortcut
            encode_ids[i] = -1;
            record.s_rev[i] = 1.0 / coef;
        } else {
            // real fft
            int id = fft_count;
            ++fft_count;
            encode_ids[i] = id;
            decode_ids[id] = i;
        }
    }

    record.s_size = s_size;
    if(record.x_dim == x_dim && record.fft_count == fft_count) {
        // well done
        // do nothing
    } else {
        if(record.dev_damp || record.dev_origin || record.dev_freq) {
            assert(false);
            hipFree(record.dev_damp);
            hipFree(record.dev_origin);
            hipFree(record.dev_freq);
            hipFree(record.dev_inout);
            hipfftDestroy(record.fwd_plan);
            hipfftDestroy(record.bck_plan);
        }
        record.x_dim = x_dim;
        record.fft_count = fft_count;
        record.dev_damp = cuda_alloc<double>(fft_count * (x_dim + 2));
        record.dev_origin = cuda_alloc<double>(fft_count * x_dim);
        record.dev_freq = cuda_alloc<hipfftDoubleComplex>(fft_count * (x_dim + 2) / 2);
        record.dev_inout = cuda_alloc<double>(s_size * x_dim);

        hipfftPlan1d(&record.fwd_plan, x_dim, HIPFFT_R2C, fft_count);
        hipfftPlan1d(&record.bck_plan, x_dim, HIPFFT_C2R, fft_count);
    }
    double placeholder[4] = {1.0, 1.0, 1.0, 1.0};
    for(int id = 0; id < fft_count; id++) {
        int i = decode_ids[id];
        double* ptr = record.dev_damp + id * (x_dim + 2);
        // set damp
        hipMemcpy(ptr, placeholder, sizeof(placeholder), hipMemcpyHostToDevice);
        hipMemcpy(ptr + 4, damp_ + 2, sizeof(double) * (x_dim - 2),
                   hipMemcpyHostToDevice);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(dev_pft_records), pft_records + plan_id, sizeof(PftRecord),
                       sizeof(PftRecord) * plan_id);
}

__global__ void pft_prepare(double* __restrict__ p_inout, int plan_id) {
    int s_index = blockIdx.x;
    int x_id = threadIdx.x;
    auto& record = dev_pft_records[plan_id];
    int x_dim = record.x_dim;
    int id = record.encode_ids[s_index];
    double* raw_p = p_inout + s_index * x_dim;
    if(id == -2 || x_id >= x_dim) {
        // do nothing
    } else if(-1) {
        // inplace filter
        double s_rev = record.s_rev[s_index];
        double mid = raw_p[x_id];
        double left = x_id - 1 >= 0 ? raw_p[x_id - 1] : raw_p[x_dim];
        double right = x_id + 1 < x_dim ? raw_p[x_id + 1] : raw_p[0];
        double result = mid * s_rev + (1 - s_rev) * 0.5 * (left + right);
        __syncthreads();
        raw_p[x_id] = result;
    } else {
        // fft
        int fft_id = id;
        // fill into destination
        double* dest = record.dev_origin + fft_id * x_dim;
        dest[x_id] = raw_p[x_id];
    }
}

__global__ void pft_finish(double* __restrict__ p_inout, int plan_id) {
    int fft_id = blockIdx.x;
    int x_id = threadIdx.x;
    auto& record = dev_pft_records[plan_id];
    int x_dim = record.x_dim;
    int s_index = record.decode_ids[fft_id];
    double* src = record.dev_origin + fft_id * x_dim;
    if(x_id < x_dim) {
        p_inout[x_id] = src[x_id];
    }
}

extern "C" void cuda_pft2d_(int* plan_id_,
                            double* p_inout_    // array filtered [y_dim][x_dim]
) {
    int plan_id = *plan_id_;
    auto& record = pft_records[plan_id];
    int s_size = record.s_size;
    int x_dim = record.x_dim;
    int fft_count = record.fft_count;
    auto* dev_damp = record.dev_damp;
    auto* dev_origin = record.dev_origin;
    auto* dev_freq = record.dev_freq;
    auto* dev_inout = record.dev_inout;
    hipMemcpy(dev_inout, p_inout_, sizeof(double) * s_size * x_dim,
               hipMemcpyHostToDevice);
    // may change to benifit the hardware
    pft_prepare<<<s_size, x_dim>>>(dev_inout, plan_id);
    hipfftExecD2Z(record.fwd_plan, dev_origin, dev_freq);
    thrust::transform(thrust::system::cuda::par,
                      (double*)dev_freq,                              //
                      (double*)dev_freq + fft_count * (x_dim + 2),    //
                      dev_damp,                                       //
                      (double*)dev_freq,                              //
                      [] __device__(double a, double b) { return a * b; });
    hipfftExecZ2D(record.bck_plan, dev_freq, dev_origin);
    pft_finish < <<s_size, x_dim>>(dev_inout, plan_id);
    hipMemcpy(p_inout_, dev_inout, sizeof(double) * s_size * x_dim,
               hipMemcpyDeviceToHost);
}
