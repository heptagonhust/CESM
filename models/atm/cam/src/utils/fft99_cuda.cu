//只是放在这里，还没有研究fft的输入输出
//如果需要单独把 init 函数和 destroy 函数抽出来，我再到调用的地方加
//https://github.com/qzwlecr/CESM/issues/12
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <fstream>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>
#include <string>
#include <unordered_map>

#include "fft99_cuda.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
using namespace std::chrono;

#define LOG(arg) printf("%s=%d, ", #arg, (arg))

using real_t = hipfftDoubleReal;
using complex_t = hipfftDoubleComplex;
extern "C" void cuda_fft991_batch_host_(       //
    int* batch_size_, int* batch_distance_,    //
    double* a_,                                // inout, elements[lot][N+2]
    int* inc_,                                 // data memory addr increment of elements
    int* jump_,                                // data memory addr increment of vector

    int* n_,                                   // count of elements in a vector
    int* lot_,                                 // count of vectors
    int* ISIGN_                                // -1 => time2freq, +1 => freq2time
) {
    // to be discard
    auto batch_size = *batch_size_;
    auto batch_distance = *batch_distance_;
    double* dev_a;
    int data_byte = batch_distance * batch_size * sizeof(double);
    hipMalloc(&dev_a, data_byte);
    hipMemcpy(dev_a, a_, data_byte, hipMemcpyHostToDevice);
    if(batch_distance == *jump_ * *lot_) {
        int fake_lot = *lot_ * batch_size;
        cuda_fft991_(dev_a, inc_, jump_, n_, &fake_lot, ISIGN_);
    } else {
        assert(false);
    }
    hipMemcpy(a_, dev_a, data_byte, hipMemcpyDeviceToHost);
    hipFree(dev_a);
}

extern "C" void cuda_fft991_(    //
    double* a_,                  // inout, elements[lot][N+2]
    int* inc_,                   // data memory addr increment of elements
    int* jump_,                  // data memory addr increment of vector
    int* n_,                     // count of elements in a vector
    int* lot_,                   // count of vectors
    int* ISIGN_                  // -1 => time2freq, +1 => freq2time

) {
    // assume
    // thread_local hipfftHandle fwd_plan, bck_plan;
    // thread_local int total_count = 0;
    // thread_local bool init_flag = false;
    // thread_local int batch_count = 0;
    // if(!init_flag || batch_count != *lot_) {
    //     if(init_flag) {
    //         hipfftDestroy(fwd_plan);
    //         hipfftDestroy(bck_plan);
    //         init_flag = false;
    //     }
    //     int n = *n_;
    //     int stride = *inc_;
    //     int real_dist = *jump_;
    //     int complex_dist = real_dist / 2;
    //     int ranks[] = {n};
    //     batch_count = *lot_;
    //     int new_count = batch_count * real_dist;
    //     // assert(!total_count || new_count == total_count);
    //     total_count = new_count;
    //     printf("\n<<<<");
    //     LOG(n);
    //     LOG(stride);
    //     LOG(real_dist);
    //     LOG(batch_count);
    //     printf("\n");
    //     hipfftPlanMany(&fwd_plan, 1, ranks, nullptr, stride, real_dist, nullptr, stride,
    //                   complex_dist, HIPFFT_D2Z, batch_count);
    //     hipfftPlanMany(&bck_plan, 1, ranks, nullptr, stride, complex_dist, nullptr, stride,
    //                   real_dist, HIPFFT_Z2D, batch_count);
    //     init_flag = true;
    // }
    thread_local std::unordered_map<int, std::pair<hipfftHandle, hipfftHandle> > record;
    int batch_count = *lot_;
    assert(*jump_ == 146);
    int total_count = 146 * batch_count;
    if(record.count(batch_count) == 0){
        auto& fwd_plan = record[batch_count].first;
        auto& bck_plan = record[batch_count].second;
        int n = *n_;
        int stride = *inc_;
        int real_dist = *jump_;
        int complex_dist = real_dist / 2;
        int ranks[] = {n};
        int new_count = batch_count * real_dist;
        // assert(!total_count || new_count == total_count);
        total_count = new_count;
        printf("\n<<<<");
        LOG(n);
        LOG(stride);
        LOG(real_dist);
        LOG(batch_count);
        printf("\n");
        hipfftPlanMany(&fwd_plan, 1, ranks, nullptr, stride, real_dist, nullptr, stride,
                      complex_dist, HIPFFT_D2Z, batch_count);
        hipfftPlanMany(&bck_plan, 1, ranks, nullptr, stride, complex_dist, nullptr, stride,
                      real_dist, HIPFFT_Z2D, batch_count);
    } 

    auto& fwd_plan = record[batch_count].first;
    auto& bck_plan = record[batch_count].second;
    if(*ISIGN_ == -1) {
        // fwd_plan
        auto status = hipfftExecD2Z(fwd_plan, (real_t*)a_, (complex_t*)a_);
        // std::cout << "total_count=" << total_count << std::endl;
        // auto fn =;
        thrust::transform(thrust::system::cuda::par, a_, a_ + total_count, a_,
                          [=] __device__(double x) { return x / 144; });
        // printf("{executed=%d}", status);
    } else {
        hipfftExecZ2D(bck_plan, (complex_t*)a_, (real_t*)a_);
    }
}

extern "C" void needle_(                      //
    double* a_,                               // inout, elements[lot][N+2]
    int* batch_size_, int* batch_distance_    // distance
) {
    static int counter = 0;
    printf("[needle at %p, size=%d, dist=%d]", a_, *batch_size_, *batch_distance_);
    ++counter;
    auto current_time = system_clock::now();
    auto path = "/home/mike/tmp/";
    auto filename = path + std::to_string(system_clock::to_time_t(current_time)) + "-" +
                    std::to_string(counter) + ".dat";
    int data_byte = *batch_distance_ * *batch_size_ * sizeof(double);
    std::ofstream fout(filename, std::ios::binary);
    fout.write((char*)a_, data_byte);
}
