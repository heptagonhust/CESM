#include "hip/hip_runtime.h"
//只是放在这里，还没有研究fft的输入输出
//如果需要单独把 init 函数和 destroy 函数抽出来，我再到调用的地方加
//https://github.com/qzwlecr/CESM/issues/12
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <fstream>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
using namespace std::chrono;

extern "C" void needle_(                      //
    double* a_,                               // inout, elements[lot][N+2]
    int* batch_size_, int* batch_distance_    // distance
) {
    static int counter = 0;
    printf("[needle at %p, size=%d, dist=%d]", a_, *batch_size_, *batch_distance_);
    ++counter;
    auto current_time = system_clock::now();
    auto path = "/home/mike/tmp/";
    auto filename = path + std::to_string(system_clock::to_time_t(current_time)) + "-" +
                    std::to_string(counter) + ".dat";
    int data_byte = *batch_distance_ * *batch_size_ * sizeof(double);
    std::ofstream fout(filename, std::ios::binary);
    fout.write((char*)a_, data_byte);
}

template <typename T, bool is_managed = false>
T* cuda_alloc(int size) {
    int bytes = sizeof(T) * size;
    T* tmp;
    if(is_managed) {
        hipMallocManaged(&tmp, bytes);
    } else {
        hipMalloc(&tmp, bytes);
    }
    return tmp;
}

// blocks: y_dim
// threads: x_dim
// s_ should be constant array

// need adjustment
constexpr int MAX_S_SIZE = 128 + 16;
struct PftRecord {
    // here is the wtf
    int s_size;
    int x_dim;
    double s_rev[MAX_S_SIZE];
    int fft_count;
    int encode_ids[MAX_S_SIZE];
    int decode_ids[MAX_S_SIZE];
    int fwd_plan, bck_plan;
    double* dev_damp;                // of fft_count * (x_dim + 2), keep it in memory
    hipfftDoubleReal* dev_origin;     // of fft_count * (x_dim), keep it in memory
    hipfftDoubleComplex* dev_freq;    // of fft_count * (x_dim + 2), keep it in memory
    double* dev_inout;               // of s_size * x_dim
};

static PftRecord pft_records[4] = {};
static __constant__ PftRecord dev_pft_records[4];

extern "C"    //
    void
    cuda_pft_cf_record_(int* plan_id_, double* s_, int* s_beg_, int* s_end_,
                        double* damp_, int* im_, int* fft_flt_) {
    // initalize this region for further call
    int plan_id = *plan_id_;
    assert(plan_id >= 0 && plan_id < 4);
    int s_size = *s_end_ - *s_beg_ + 1;
    assert(s_size < MAX_S_SIZE);
    int x_dim = *im_;
    auto& record = pft_records[plan_id];
    auto* encode_ids = record.encode_ids;
    auto* decode_ids = record.decode_ids;

    int fft_count = 0;

    bool force_fft = (bool)*fft_flt_;
    printf("{plan_id=%d}", plan_id);
    for(int i = 0; i < s_size; ++i) {
        auto coef = s_[i];
        printf("<%ld>", coef);
        record.s_rev[i] = 1.0 / coef;
        if(coef <= 1.01) {
            // skip
            encode_ids[i] = -2;
        } else if(!force_fft && coef <= 4.0) {
            // shortcut
            encode_ids[i] = -1;
        } else {
            // real fft
            int id = fft_count;
            ++fft_count;
            encode_ids[i] = id;
            decode_ids[id] = i;
        }
    }
    printf("\n");
    record.s_size = s_size;
    if(record.x_dim == x_dim && record.fft_count == fft_count) {
        // well done
        // do nothing
    } else {
        if(record.dev_damp || record.dev_origin || record.dev_freq) {
            assert(false);
            hipFree(record.dev_damp);
            hipFree(record.dev_origin);
            hipFree(record.dev_freq);
            hipFree(record.dev_inout);
            hipfftDestroy(record.fwd_plan);
            hipfftDestroy(record.bck_plan);
        }
        record.x_dim = x_dim;
        record.fft_count = fft_count;
        record.dev_damp = cuda_alloc<double>(fft_count * (x_dim + 2));
        record.dev_origin = cuda_alloc<double>(fft_count * x_dim);
        record.dev_freq = cuda_alloc<hipfftDoubleComplex>(fft_count * (x_dim + 2) / 2);
        record.dev_inout = cuda_alloc<double>(s_size * x_dim);

        hipfftPlan1d(&record.fwd_plan, x_dim, HIPFFT_R2C, fft_count);
        hipfftPlan1d(&record.bck_plan, x_dim, HIPFFT_C2R, fft_count);
    }
    double placeholder[4] = {1.0, 1.0, 1.0, 1.0};
    for(int id = 0; id < fft_count; id++) {
        int i = decode_ids[id];
        double* dev_damp_ptr = record.dev_damp + id * (x_dim + 2);
        double* host_damp_ptr = damp_ + i * x_dim;
        // set damp
        hipMemcpy(dev_damp_ptr, placeholder, sizeof(placeholder),
                   hipMemcpyHostToDevice);
        hipMemcpy(dev_damp_ptr + 4, host_damp_ptr + 2, sizeof(double) * (x_dim - 2),
                   hipMemcpyHostToDevice);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(dev_pft_records), pft_records + plan_id, sizeof(PftRecord),
                       sizeof(PftRecord) * plan_id);
}

__global__ void pft_prepare(double* __restrict__ p_inout, int plan_id) {
    int s_index = blockIdx.x;
    int x_id = threadIdx.x;
    auto& record = dev_pft_records[plan_id];
    int x_dim = record.x_dim;
    int id = record.encode_ids[s_index];
    double* raw_p = p_inout + s_index * x_dim;
    if(id == -2 || x_id >= x_dim) {
        // do nothing
    } else if(-1) {
        // inplace filter
        double s_rev = record.s_rev[s_index];
        double mid = raw_p[x_id];
        double left = x_id - 1 >= 0 ? raw_p[x_id - 1] : raw_p[x_dim];
        double right = x_id + 1 < x_dim ? raw_p[x_id + 1] : raw_p[0];
        double result = mid * s_rev + (1 - s_rev) * 0.5 * (left + right);
        __syncthreads();
        raw_p[x_id] = result;
    } else {
        // fft
        int fft_id = id;
        // fill into destination
        double* dest = record.dev_origin + fft_id * x_dim;
        dest[x_id] = raw_p[x_id];
    }
}

__global__ void pft_finish(double* __restrict__ p_inout, int plan_id) {
    int fft_id = blockIdx.x;
    auto& record = dev_pft_records[plan_id];
    int x_dim = record.x_dim;
    int s_index = record.decode_ids[fft_id];
    double* src = record.dev_origin + fft_id * x_dim;
    double* dest = p_inout + s_index * x_dim;

    int x_id = threadIdx.x;
    if(x_id < x_dim) {
        dest[x_id] = src[x_id];
    }
}

extern "C" void cuda_pft2d_(double* p_inout_,    // array filtered [y_dim][x_dim]
                            int* plan_id_, //
                            // raw datas
                            double* xxx_s, double* xxx_d, //
                            int* xxx_im, int* xxx_jp//
                            ) {
    int plan_id = *plan_id_;
    auto& record = pft_records[plan_id];
    int s_size = record.s_size;
    int x_dim = record.x_dim;
    int fft_count = record.fft_count;
    auto* dev_damp = record.dev_damp;
    auto* dev_origin = record.dev_origin;
    auto* dev_freq = record.dev_freq;
    auto* dev_inout = record.dev_inout;
    assert(*xxx_im == x_dim);
    assert(*xxx_jp == s_size);
    double wtf = xxx_s[14] - 1.0 / record.s_rev[14];
    assert((float)(wtf) == (float)0.0);
    // what about d?  
    hipMemcpy(dev_inout, p_inout_, sizeof(double) * s_size * x_dim,
               hipMemcpyHostToDevice);
    // may change to benifit the hardware
    pft_prepare<<<s_size, x_dim>>>(dev_inout, plan_id);
    hipfftExecD2Z(record.fwd_plan, dev_origin, dev_freq);
    thrust::transform(thrust::system::cuda::par,
                      (double*)dev_freq,                              //
                      (double*)dev_freq + fft_count * (x_dim + 2),    //
                      dev_damp,                                       //
                      (double*)dev_freq,                              //
                      [] __device__(double a, double b) { return a * b / 144.0; });
    hipfftExecZ2D(record.bck_plan, dev_freq, dev_origin);
    pft_finish<<<s_size, x_dim>>>(dev_inout, plan_id);
    hipMemcpy(p_inout_, dev_inout, sizeof(double) * s_size * x_dim,
               hipMemcpyDeviceToHost);
}
